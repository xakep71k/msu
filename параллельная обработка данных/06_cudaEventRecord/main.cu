#include <hip/hip_runtime.h>
//
// Программа фиксирует два события с помощью cudaEventRecord и выводит время между ними
//

__global__ void kernel() {}

int main()
{
    // Создаем события
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    // Фиксируем события до и после запуска kernel
    hipEventRecord(event1, 0); // 0 - поток по умолчанию
    kernel<<<1,1>>>();
    hipEventRecord(event2, 0);

    // Синхронизируем события
    hipEventSynchronize(event1); // Не обязательно
    hipEventSynchronize(event2); // Дожидаеся когда будет выполнено

    // Вычисляем время между cudaEventRecord
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);

    return 0;
}
