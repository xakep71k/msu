#include <hip/hip_runtime.h>
#include <stdio.h>
//
// Программа фиксирует два события с помощью cudaEventRecord и выводит время между ними
//

__global__ void kernel() {}

int main()
{
    // Создаем события
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    // Фиксируем события до и после запуска kernel
    hipEventRecord(event1, 0); // 0 - поток по умолчанию
    dim3 grid(3,2,1), block(4,3,1);
    kernel<<<grid, block>>>();
    hipEventRecord(event2, 0);

    // Синхронизируем события
    hipEventSynchronize(event1); // Не обязательно
    hipEventSynchronize(event2); // Дожидаеся когда будет выполнено

    // Вычисляем время между cudaEventRecord
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("%f\n", dt_ms);

    return 0;
}
